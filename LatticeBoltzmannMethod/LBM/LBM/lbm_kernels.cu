#include "hip/hip_runtime.h"
#pragma once
#include "lbm_kernels.cuh"

#include <stdio.h>

__global__ void lbm_calc(LBM::LatticeInfo* l_info,LBM::LBMPoint* points){
	//�X���b�hID�ƃu���b�NID����C���f�b�N�X���Z�o
	int index_in = blockIdx.x * blockDim.x + threadIdx.x;
	
	//���E�𒴂���ꍇ��0
	int branch = index_in >= 0;
	branch *= index_in < l_info->size;

	//�Q�Ɨ̈���܂߂��C���f�b�N�X���Z�o
	int index_out = l_info->offset 
		+ index_in 
		+ (index_in / ( l_info->x_max_in * l_info->y_max_in ))
			* (l_info->x_max_out * l_info->y_max_out - l_info->x_max_in * l_info->y_max_in) 
		+ (index_in / l_info->x_max_in) % 2 
			* l_info->max_speed
			* 2;
	index_out = index_out * branch;

	printf("lbm_calc in : %d  out : %d size : %d branch : %d\n",index_in,index_out,l_info->size,branch);

	//�v�Z
	points[index_out].density = 0;
	points[index_out].vx = 0;
	points[index_out].vy = 0;
	points[index_out].vz = 0;

	for(int n = 0;n < l_info->direct_num;n++){
		printf("	access a : %d index : %d\n",n,index_in);
		points[index_out].a[n] = 0;
	}
}

__global__ void lbm_test1(){
	int index = threadIdx.x;

	printf("test1 %d\n",index);
}

__global__ void lbm_test2(){
	int index = threadIdx.x;

	printf("test2 %d\n",index);
}