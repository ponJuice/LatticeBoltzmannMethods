#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""
#include "lbm_kernels.cuh"
#include <stdio.h>
#include <math.h>

#include "lbm_data.h"
#include "lbm_util.h"

int main(){
	hipError_t ce;

	int device_num = 0;	//TITAN���g�p
	hipDeviceProp_t dev;
	hipGetDeviceProperties(&dev,device_num);

	//�v�Z�̈�̏��
	LBM::LatticeInfo info;
	LBM::getLbmLatticeInfo(&info,1,4,4,4,15);
	LBM::printLatticeInfo(&info);

	int threads = Math::min(dev.maxThreadsPerBlock,info.size);//32 * ((info.size - 1) / 32 + 1);
	int block = (int)ceil((double)threads/(double)dev.maxThreadsPerBlock);

	printf("threads : %d\n",threads);
	printf("block : %d\n",block);

	int a_array_size = sizeof(double) * info.direct_num;
	int point_size = 36;//sizeof(LBM::LBMPoint);

	int a = sizeof(double*);

	printf("a_size : %d\n",a_array_size);
	printf("point size : %d\n",point_size);

	LBM::LBMPoint* d_point;
	LBM::LBMPoint h_point;
	LBM::LatticeInfo* d_info;

	ce = hipMalloc((void**)&d_point,point_size*info.x_max_out*info.y_max_out*info.z_max_out);
	if(ce != hipSuccess){
		printf("failed hipMalloc <<point>>\n");
		printf("error : %s\n",hipGetErrorString(ce));
		goto allCudaFree;
	}
	for(int n = 0;n < info.size;n++){
		double* _a;
		ce = hipMalloc((void**)&_a,a_array_size);
		//ce = hipMalloc((void**)&d_point[n].a,a_size * info.direct_num);
		if(ce != hipSuccess){
			printf("failed hipMalloc <<point.a>>\n");
			printf("error : %s\n",hipGetErrorString(ce));
			goto allCudaFree;
		}
		h_point.a = _a;
		d_point[n].a = _a;
		ce = hipMemcpy(&d_point[n],&h_point,point_size,hipMemcpyHostToDevice);
		if(ce != hipSuccess){
			printf("failed hipMemcpy <<point.a>>\n");
			printf("error : %s\n",hipGetErrorString(ce));
			goto allCudaFree;
		}
	}

	int info_size = sizeof(LBM::LatticeInfo);
	ce = hipMalloc((void**)&d_info,info_size);
	if(ce != hipSuccess){
		printf("failed hipMalloc <<info>>\n");
		printf("error : %s\n",hipGetErrorString(ce));
		goto allCudaFree;
	}

	ce = hipMemcpy(d_info,&info,info_size,hipMemcpyHostToDevice);	
	if(ce != hipSuccess){
		printf("failed hipMemcpy <<info>>\n");
		printf("error : %s\n",hipGetErrorString(ce));
		goto allCudaFree;
	}

	lbm_calc<<<block,threads>>>(d_info,d_point);
	hipDeviceSynchronize();
	ce = hipGetLastError();
	if(ce != hipSuccess){
		printf("failed karnel <<lbm_calc>>\n");
		printf("error : %s\n",hipGetErrorString(ce));
		goto allCudaFree;
	}
	

allCudaFree:
	for(int n = 0;n < info.size;n++){
		ce = hipFree(d_point[n].a);
		if(ce != hipSuccess){
			printf("failed hipFree <<point.a>>\n");
			printf("error : %s\n",hipGetErrorString(ce));
		}
	}
	ce = hipFree(d_point);
	if(ce != hipSuccess){
		printf("failed hipFree <<point>>\n");
		printf("error : %s\n",hipGetErrorString(ce));
	}
	ce = hipFree(d_info);
	if(ce != hipSuccess){
		printf("failed hipFree <<info>>\n");
		printf("error : %s\n",hipGetErrorString(ce));
	}
	hipDeviceReset();

	return 0;
}